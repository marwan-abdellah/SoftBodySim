#include "hip/hip_runtime.h"
#include "glm/glm.hpp"
#include "CUDASoftBodySolverKernel.h"
#include "common.h"

using namespace glm;

__device__ uint_t hash(uint_t id)
{
	return 1193 * id;
}

__global__ void cudaUpdateVelocitiesKernel(
		vec3 gravity,
		vec3 *positions,
		vec3 *projections,
		vec3 *velocities,
		vec3 *ext_forces,
		float_t *masses,
		float_t dt,
		uint_t max_idx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx < max_idx) {

		// 0. Load from global mem.
		vec3 position = positions[idx];
		vec3 force = ext_forces[idx];
		float_t imass = masses[idx];
		vec3 velocity = velocities[idx];
		
		// 1. Updating velocities.
		velocity += dt * imass * (force + gravity);

		// 2. Damp velocities.
		velocity *= 0.99f; // Naive damping

		// 3. projecting positions
		vec3 projection = position + velocity * dt;

		// update global tables
		projections[idx] = projection;
		velocities[idx] = velocity;
	}
}

/**
  step 4. solving links constraints.
  */
__global__ void solveConstraints(
		unsigned int max_steps,
		LinkConstraint *links,
		glm::vec3 *projections,
		glm::float_t *masses_inv,
		glm::uint_t max_idx)
{
	__shared__ vec3   ACCUM[2 * MAX_LINKS];
	__shared__ uint_t COUNTER[2 * MAX_LINKS];

	int link_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (link_idx < max_idx) {

		LinkConstraint lnk = links[link_idx];
		glm::vec3 pos0 = projections[lnk.index[0]];
		glm::vec3 pos1 = projections[lnk.index[1]];
		glm::float_t mass_inv0 = masses_inv[lnk.index[0]];
		glm::float_t mass_inv1 = masses_inv[lnk.index[1]];

		// assume that will be no colliosions; MAX_LINS = 2^x, X in N
		uint_t id0 = hash(lnk.index[0]) & (2 * MAX_LINKS - 1);
		uint_t id1 = hash(lnk.index[1]) & (2 * MAX_LINKS - 1);

		ACCUM[id0] = pos0;
		ACCUM[id1] = pos1;
		COUNTER[id0] = 1;
		COUNTER[id1] = 1;

		__syncthreads();

		glm::float_t restLen = lnk.restLength;
		glm::float_t k = lnk.stiffness;

		glm::vec3 diff = pos0 - pos1;
		glm::float_t len = length(diff);

		float_t m0 = mass_inv0 / (mass_inv0 + mass_inv1) * (len - restLen) /
			len;
		float_t m1 = mass_inv1 / (mass_inv0 + mass_inv1) * (len - restLen) /
			len;

		pos0 -= k * m0 * diff;
		pos1 += k * m1 * diff;

		atomicAdd(&ACCUM[id0][0], pos0[0]);
		atomicAdd(&ACCUM[id0][1], pos0[1]);
		atomicAdd(&ACCUM[id0][2], pos0[2]);
		atomicAdd(&ACCUM[id1][0], pos1[0]);
		atomicAdd(&ACCUM[id1][1], pos1[1]);
		atomicAdd(&ACCUM[id1][2], pos1[2]);

		atomicInc(&COUNTER[id0], MAX_LINKS);
		atomicInc(&COUNTER[id1], MAX_LINKS);

		__syncthreads();

		pos0 = ACCUM[id0] * (1.0f / (float_t)COUNTER[id0]);
		pos1 = ACCUM[id1] * (1.0f / (float_t)COUNTER[id1]);

		projections[lnk.index[0]] = pos0;
		projections[lnk.index[1]] = pos1;
	}
}


/**
  step 5. solving collision constraints.
  */
__global__ void solvePointTriangleCollisionsKernel(
		CollisionPointTriangleConstraint2 *collisions_data,
		uint_t max_idx
	)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		CollisionPointTriangleConstraint2 cd = collisions_data[idx];

		vec3 position = cd.positions[cd.pointIdx];
		vec3 projection = cd.projections[cd.pointIdx];

		vec3 tri0 = cd.trianglePositions[cd.triangleIdxs[0]];
		vec3 tri1 = cd.trianglePositions[cd.triangleIdxs[1]];
		vec3 tri2 = cd.trianglePositions[cd.triangleIdxs[2]];

		vec3 a = tri1 - tri0;
		vec3 b = tri2 - tri0;

		vec3 norm = cross(a, b);
		vec3 diff = projection - position;
		float_t k = dot(norm, diff);
		if (k < 0.0001f)
			return;

		// calculate plane intersection point
		k = dot(norm, (tri1 - position)) / k;
		vec3 q = k * (position + diff);
		
		k = dot(norm, projection - q);
		if (k > 0.01f)
			return;

		cd.projections[cd.pointIdx] = q;
	}
}

//
///**
//  */
//
/**
  step 6. Integrate motion.
  */
__global__ void integrateMotionKernel(
		glm::float_t dt,
		glm::vec3 *positions,
		glm::vec3 *projections,
		glm::vec3 *velocities,
		glm::uint max_idx
		)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		glm::vec3 pos = positions[idx];
		glm::vec3 proj = projections[idx];

		velocities[idx] = (proj - pos) / dt;
		positions[idx] = proj;
	}
}

__global__ void cudaUpdateVertexBufferKernel(Vertex *vboPtr, glm::vec3 *positions, glm::uint *mapping, glm::uint max_idx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		glm::uint index = mapping[idx];
		glm::vec3 vertex = positions[index];
		vboPtr[idx].position = vertex;
	}
}

__global__ void calculateLinkStiffness(
		unsigned int solver_steps,
		LinkConstraint *links,
		glm::uint_t max_idx)
{
	int link_idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (link_idx < max_idx) {
		LinkConstraint lnk = links[link_idx];

		links[link_idx].stiffness = 1.0f - powf(1.0 - lnk.stiffness, 1.0f /
				solver_steps);
	}
}
