#include "CUDASoftBodySolver.h"

#include "common.h"
#include <cstring>

using namespace std;
using namespace glm;

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include "CUDASoftBodySolverKernel.h"

#define DEFAULT_SOLVER_STEPS 10

struct CUDASoftBodySolver::CollisionBodyInfoDescriptor {
	vec3              *positions;
	CollisionBodyInfo collInfo;
};

struct CUDASoftBodySolver::SoftBodyDescriptor {
	SoftBody             *body;
	hipGraphicsResource *graphics;

	vec3                 *positions;
	vec3                 *projections;
	vec3                 *velocities;
	vec3                 *forces;
	float_t              *massesInv;
	unsigned int         nParticles;
	LinkConstraint       *links;
	unsigned int         nLinks;
	uint_t               *mapping;  /* Mapping between particles positions and vertexes 
									   is VertexBuffer.
									 Used for updating Vertex poistions */
	unsigned int         nMapping;
};

struct CUDASoftBodySolver::SolverPrivate {
	int             deviceId;
	hipDeviceProp_t  devProp;
	hipStream_t	stream;
	int             solverSteps;

	descriptorArray_t             descriptors;
	vector<hipGraphicsResource*> resArray; /* helper array to map all resources in one call */

	collisionBodyDescriptorArray_t  collBodyDescrHost;
	CollisionBodyInfoDescriptor     *collBodyDescrDevice;

	CollisionPointTriangleConstraint2  *collisions;
	unsigned int                        nCollisions;
};

CUDASoftBodySolver::CUDASoftBodySolver(SimulationType type)
	:
		mCuda(0),
		mInitialized(false),
		mGravity(0, -10.0f, 0),
		mType(type)
{
}

CUDASoftBodySolver::~CUDASoftBodySolver(void)
{
	shutdown();
}

bool CUDASoftBodySolver::cudaInitializeDevice(SolverPrivate *cuda)
{
	hipError_t err;
	hipDeviceProp_t  prop;
	memset(&prop, 0x0, sizeof(prop));
	prop.major = 3;
	prop.minor = 5;

	// choose device for us. Prefer with compute capabilities ~ 3.5
	err = hipChooseDevice(&cuda->deviceId, &prop);
	if (err != hipSuccess) return false;

	err = hipSetDevice(cuda->deviceId);
	if (err != hipSuccess) return false;

	err = hipGetDeviceProperties(&cuda->devProp, cuda->deviceId);
	if (err != hipSuccess) return false;
	
	err = hipStreamCreate(&cuda->stream);
	if (err != hipSuccess) return false;

	DBG("Choosen CUDA Device: %s", cuda->devProp.name);
	DBG("Multiprocessor count: %d", cuda->devProp.multiProcessorCount);
	DBG("Compute capability: %d.%d", cuda->devProp.major, cuda->devProp.minor);
	DBG("Total global mem: %d", cuda->devProp.totalGlobalMem);
	DBG("Shared mem per block: %d", cuda->devProp.sharedMemPerBlock);
	DBG("Warp size: %d", cuda->devProp.warpSize);
	DBG("Concurrent kenels: %d", cuda->devProp.concurrentKernels);

	return true;
}

bool CUDASoftBodySolver::cudaShutdownDevice(SolverPrivate *cuda)
{
	hipError_t err;

	err = hipDeviceSynchronize();
	if (err != hipSuccess) return false;

	err = hipDeviceReset();
	if (err != hipSuccess) return false;

	return true;
}

static void *allocateCUDABuffer(size_t bytes, bool zeroed=false)
{
	hipError_t err;
	void *ret = NULL;
	err = hipMalloc(&ret, bytes);
	if (err != hipSuccess) {
		ERR("%s", hipGetErrorString(err));
		return NULL;
	}
	if (zeroed) {
		err = hipMemset(ret, 0x0, bytes);
		if (err != hipSuccess) {
			ERR("%s", hipGetErrorString(err));
			return NULL;
		}
	}
	return ret;
}

long CUDASoftBodySolver::cudaAllocateDeviceBuffers(SoftBodyDescriptor *descr)
{
	int bytesArray = 0, bytesMass = 0, bytesMapping = 0, bytesLinks = 0;

	bytesArray = descr->nParticles * sizeof(vec3);
	bytesMass = descr->nParticles * sizeof(float_t);
	bytesMapping = descr->nMapping * sizeof(uint_t);
	bytesLinks = descr->nLinks * sizeof(LinkConstraint);

	descr->positions = (vec3*)allocateCUDABuffer(bytesArray);
	if (!descr->positions) goto on_fail;

	descr->projections = (vec3*)allocateCUDABuffer(bytesArray);
	if (!descr->projections) goto on_fail;

	descr->velocities = (vec3*)allocateCUDABuffer(bytesArray, true);
	if (!descr->velocities) goto on_fail;

	descr->forces  = (vec3*)allocateCUDABuffer(bytesArray, true);
	if (!descr->forces) goto on_fail;

	descr->massesInv = (float_t*)allocateCUDABuffer(bytesMass);
	if (!descr->massesInv) goto on_fail;

	descr->mapping = (uint*)allocateCUDABuffer(bytesMapping);
	if (!descr->mapping) goto on_fail;

	descr->links = (LinkConstraint*)allocateCUDABuffer(bytesLinks);
	if (!descr->links) goto on_fail;

	return 4 * bytesArray + bytesMass + bytesMapping + bytesLinks;

on_fail:
	cudaDeallocateDeviceBuffers(descr);
	return -1;
}

void CUDASoftBodySolver::cudaDeallocateDeviceBuffers(SoftBodyDescriptor *descr)
{
	if (descr->positions)   hipFree(descr->positions);
	if (descr->projections) hipFree(descr->projections);
	if (descr->velocities)  hipFree(descr->velocities);
	if (descr->forces)      hipFree(descr->forces);
	if (descr->massesInv)   hipFree(descr->massesInv);
	if (descr->links)       hipFree(descr->links);
	if (descr->mapping)     hipFree(descr->mapping);
}

CUDASoftBodySolver::SoftBodyDescriptor CUDASoftBodySolver::cudaCreateDescriptor(SoftBody *body)
{
	SoftBodyDescriptor descr;

	descr.body       = body;
	descr.graphics   = NULL;
	descr.nParticles = body->mParticles.size();
	descr.nLinks     = body->mLinks.size();
	descr.nMapping   = body->mMeshVertexParticleMapping.size();

	return descr;
}

bool CUDASoftBodySolver::cudaCopyBodyToDeviceBuffers(SoftBodyDescriptor *descr)
{
	hipError_t err;

	SoftBody *body = descr->body;

	unsigned int bytesPart = descr->nParticles * sizeof(vec3);
	unsigned int bytesLnk  = descr->nLinks * sizeof(LinkConstraint);
	unsigned int bytesMass = descr->nParticles * sizeof(float_t);
	unsigned int bytesMap  = descr->nParticles * sizeof(uint_t);

	err = hipMemcpy(descr->positions, &(body->mParticles[0]), bytesPart, hipMemcpyHostToDevice);
	if (err != hipSuccess) return false;

	err = hipMemcpy(descr->forces, &(body->mForces[0]), bytesPart, hipMemcpyHostToDevice);
	if (err != hipSuccess) return false;

	err = hipMemcpy(descr->mapping, &(body->mMeshVertexParticleMapping[0]), bytesMap, hipMemcpyHostToDevice);
	if (err != hipSuccess) return false;

	err = hipMemcpy(descr->massesInv, &(body->mMassInv[0]), bytesMass, hipMemcpyHostToDevice);
	if (err != hipSuccess) return false;

	err = hipMemcpy(descr->links, &(body->mLinks[0]), bytesLnk, hipMemcpyHostToDevice);
	if (err != hipSuccess) return false;

	return true;
}

hipGraphicsResource *cudaRegisterGLGraphicsResource(const GLVertexBuffer *vb)
{
	hipError_t err;
	hipGraphicsResource *ret = NULL;
	GLuint id = vb->getVBO(GLVertexBuffer::VERTEX_ATTR_POSITION);
	err = hipGraphicsGLRegisterBuffer(&ret, id, hipGraphicsRegisterFlagsNone);
	if (err != hipSuccess) {
		ERR("Unable to register GL buffer object %d", id);
		return false;
	}
	return ret;
}

bool CUDASoftBodySolver::cudaRegisterVertexBuffers(SoftBodyDescriptor *descr)
{
	const Mesh_t *mesh;

	if (!descr->body) {
		ERR("No SoftBody reference in descriptor!");
		return false;
	}

	mesh = descr->body->getMesh();
	if (!mesh) {
		ERR("No mesh data");
		return false;
	}

	const VertexBuffer *buf = mesh->vertexes;
	if (buf) {
		switch (buf->getType()) {
			case VertexBuffer::OPENGL_BUFFER:
				descr->graphics = cudaRegisterGLGraphicsResource(static_cast<const GLVertexBuffer*>(buf));
				if (!descr->graphics)
					return false;
				break;
			default:
				break;
		}
	}

	return true;
}

bool CUDASoftBodySolver::cudaInitCollisionDescriptors(SolverPrivate *cuda)
{
	hipError_t err;
	size_t bytes;

	if (!cuda->collBodyDescrHost.size()) return true;

	bytes = cuda->collBodyDescrHost.size() * sizeof(CollisionBodyInfoDescriptor);
	cuda->collBodyDescrDevice = (CollisionBodyInfoDescriptor*)allocateCUDABuffer(bytes);
	if (!cuda->collBodyDescrDevice) return false;

	err = hipMemcpy(cuda->collBodyDescrDevice, &cuda->collBodyDescrHost[0], bytes, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		hipFree(cuda->collBodyDescrDevice);
		cuda->collBodyDescrDevice = NULL;
		return false;
	}
	return true;
}

void CUDASoftBodySolver::cudaUpdateConstraintStiffness(SoftBodyDescriptor
		*descr, int solverSteps)
{
	int blocks = descr->nLinks / 128;
	calculateLinkStiffness<<<blocks, 128>>>(solverSteps, descr->links, descr->nLinks);
}

void CUDASoftBodySolver::cudaAppendCollsionDescriptors(collisionBodyDescriptorArray_t *arr, SoftBodyDescriptor *descr)
{
	FOREACH(it, &descr->body->mCollisionBodies) {
		CollisionBodyInfoDescriptor cb;
		cb.positions = descr->positions;
		cb.collInfo = *it;

		arr->push_back(cb);
	}
}

CUDASoftBodySolver::SolverPrivate *CUDASoftBodySolver::cudaContextCreate(softbodyArray_t *bodies)
{
	SolverPrivate *cuda;
	long total_alloc = 0;
	bool res;

	cuda = new SolverPrivate;
	memset(cuda, 0x0, sizeof(SolverPrivate));

	cuda->solverSteps = DEFAULT_SOLVER_STEPS;

	if (!cudaInitializeDevice(cuda)) {
		ERR("CUDA Device initialization failed!");
		ERR("Cuda error: %s", hipGetErrorString(hipGetLastError()));
		delete cuda;
		return NULL;
	}

	FOREACH(it, bodies) {
		if (!*it) continue;

		SoftBodyDescriptor descr = cudaCreateDescriptor(*it);

		long mem = cudaAllocateDeviceBuffers(&descr);
		if (mem == -1) {
			ERR("Unable to allocate memory for SoftBody");
			cudaContextShutdown(cuda);
			return NULL;
		}
		res = cudaCopyBodyToDeviceBuffers(&descr);
		if (!res) {
			ERR("Error occured while copying Soft bodies data to device!");
			ERR("Cuda error: %s", hipGetErrorString(hipGetLastError()));
			cudaContextShutdown(cuda);
			return NULL;
		}
		res = cudaRegisterVertexBuffers(&descr);
		if (!res) {
			ERR("Error occured registering SoftBody vertex buffers.");
			ERR("Cuda error: %s", hipGetErrorString(hipGetLastError()));
			cudaContextShutdown(cuda);
			return NULL;
		}
		cudaUpdateConstraintStiffness(&descr, cuda->solverSteps);

		cuda->descriptors.push_back(descr);
		cudaAppendCollsionDescriptors(&cuda->collBodyDescrHost, &descr);
		cuda->resArray.push_back(descr.graphics);

		total_alloc += mem;
	}
	if (!cudaInitCollisionDescriptors(cuda)) {
		ERR("Error on allocatin collision object data.");
		cudaContextShutdown(cuda);
		return NULL;
	}
	DBG("Allocated %ld bytes on device", total_alloc);

	return cuda;
}

void CUDASoftBodySolver::cudaContextShutdown(SolverPrivate *cuda)
{
	FOREACH(it, &cuda->descriptors)
		cudaDeallocateDeviceBuffers(&(*it));
	if (cuda->collBodyDescrDevice) hipFree(cuda->collBodyDescrDevice);
	cudaShutdownDevice(cuda);
	delete cuda;
}

bool CUDASoftBodySolver::initialize(softbodyArray_t *bodies)
{
	SolverPrivate *cuda;

	if (mInitialized) return true;

	cuda = cudaContextCreate(bodies);
		if (!cuda) {
		ERR("Unable to create CUDA context.");
		return false;
	}

	mInitialized = true;
	mCuda = cuda;
	return true;
}

void CUDASoftBodySolver::shutdown(void)
{
	if (!mInitialized) return;

	if (mCuda) {
		cudaContextShutdown(mCuda);
		mCuda = NULL;
	}
	mInitialized = false;
}

void CUDASoftBodySolver::updateVertexBuffers(SolverPrivate *cuda, bool async)
{
	hipError_t err;
	vec3 *ptr;
	int threadsPerBlock = 128;

	// map all in one call
	err = hipGraphicsMapResources(cuda->resArray.size(), &cuda->resArray[0]);
	if (err != hipSuccess) return;

	FOREACH(it, &cuda->descriptors) {
		size_t size;
		err = hipGraphicsResourceGetMappedPointer((void**)&ptr, &size, it->graphics);
		if (err != hipSuccess) {
			ERR("Unable to map VBO pointer");
			return;
		}
		if (size != it->nParticles * sizeof(vec3)) {
			ERR("Invalid size!");
			return;
		}
		int blockCount = it->nParticles / threadsPerBlock + 1;
		cudaUpdateVertexBufferKernel<<<blockCount, threadsPerBlock >>>(ptr,
				it->positions, it->mapping, it->nParticles);
	}

	hipGraphicsUnmapResources(cuda->resArray.size(), &cuda->resArray[0]);
}

void CUDASoftBodySolver::updateVertexBuffersAsync(void)
{
	if (mInitialized)
		updateVertexBuffers(mCuda, false); // currently only synch updates
}

void CUDASoftBodySolver::updateVertexBuffers(void)
{
	if (mInitialized)
		updateVertexBuffers(mCuda, false);
}

void CUDASoftBodySolver::projectSystem(SolverPrivate *cuda, float_t dt)
{
	int threadsPerBlock = 128;
	FOREACH(it, &cuda->descriptors) {
		int blockCount = it->nParticles / threadsPerBlock + 1;

		cudaUpdateVelocitiesKernel<<<blockCount, threadsPerBlock>>>(mGravity, it->positions,
				it->projections, it->velocities, it->forces, it->massesInv, dt,
				it->nParticles);

		threadsPerBlock = MAX_LINKS;
		blockCount = it->nLinks / threadsPerBlock + 1;

		for (int i = 0; i < cuda->solverSteps; i++)
			solveConstraints<<<blockCount, threadsPerBlock>>>(1, it->links,
					it->projections, it->massesInv, it->nLinks);

		threadsPerBlock = 128;
		blockCount = it->nParticles / threadsPerBlock + 1;
		integrateMotionKernel<<<blockCount, threadsPerBlock>>>(dt, it->positions, it->projections,
				it->velocities, it->nParticles);
//		ERR("Cuda err: %s", hipGetErrorString(hipGetLastError()));
	}
}

void CUDASoftBodySolver::projectSystem(float_t dt)
{
	if (mInitialized)
		projectSystem(mCuda, dt);
}
