#include "hip/hip_runtime.h"
#include "glm/glm.hpp"
#include "CUDASoftBodySolver.h"
#include "common.h"

///**
//  step 1. Updating velocities.
//  */
//__global__ void updateVelocities(
//		glm::vec3 gravity,
//		glm::vec3 *ext_forces,
//		glm::vec3 *velocities,
//		glm::float_t *masses,
//		glm::float_t dt,
//		glm::uint_t max_idx)
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	if ( idx < max_idx) {
//		float3 ext_force = ext_forces[idx];
//		float mass_inv = masses[idx];
//		float3 vel = velocities[idx];
//		vel += dt * ext_force * gravity * mass_inv;
//		velocities[idx] = vel;
//	}
//}
//
///**
//  step 2. Damp velocities.
//  */
//__global__ void dampVelocities(glm::vec3 *velocities, int max_idx)
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	if ( idx < max_idx) {
//		float3 vel = velocities[idx];
//		vecl *= 0.99f; // Naive damping
//		velocities[idx] = vel;
//	}
//}

/*
//  step 3. projecting positions
//  */
//__global__ void projectPositions(
//		glm::vec3 *positions,
//		glm::vec3 *projections,
//		glm::vec3 *velocities,
//		glm::float_t dt,
//		glm::uint_t max_idx)
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//	if ( idx < max_idx) {
//		float3 position = positions[idx];
//		float3 velocity = velocities[idx];
//
//		float3 projection = position + velocity * dt;
//		projections[idx] = projection;
//	}
//}
//
///**
//  step 4. solving links constraints.
//  */
//__global__ void solveLinks(
//		glm::float_t k,
//		glm::uvec2 *links,
//		glm::vec3 *projections,
//		glm::float_t *masses_inv,
//		glm::float_t *rest_length2,
//		glm::uint_t max_idx)
//{
//	int link_idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (link_idx < max_idx) {
//		glm::float_t restLen2 = rest_length2[link_idx];
//		glm::uvec2 idx = links[link_idx];
//
//		glm::vec3 pos0 = projections[idx[0]];
//		glm::vec3 pos1 = projections[idx[1]];
//
//		glm::float_t mass_inv0 = masses_inv[idx[0]];
//		glm::float_t mass_inv1 = masses_inv[idx[1]];
//
//		glm::vec3 dist = pos0 - pos1;
//		glm::float_t len2 = glm::dot(dist, dist);
//		glm::float_t c = k * (restLen2 - len2);
//
//		pos0 = pos0 - c * mass_inv0 * dist;
//		pos1 = pos1 + c * mass_inv1 * dist;
//
//		projections[idx[0]] = pos0;
//		projections[idx[1]] = pos1;
//	}
//}
//
//struct {
//	enum Type {
//		TRIANGLE,
//	} type;
//	union {
//		struct {
//			glm::uvec3 idx;
//		} triangle;
//	} data;
//	bool fixed;
//} CollisionBodyData;
//
///**
//  step 5. solving collision constraints.
//  */
//__global__ void solveCollisions(
//		glm::uvec2 *collisions,
//		CollisionBody *collisions_bodies_data,
//		glm::vec3 *projections,
//		glm::vec3 *positions,
//		glm::vec3 *velocities,
//		glm::vec3 *masses_inv,
//		glm::vec3 *forces,
//		glm::uint max_idx
//	)
//{
//	int coll_idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (coll_idx < max_idx) {
//		uvec2 *coll_data = collisions[coll_idx];
//
//		/**
//		  * coll_data[0] keeps id of colliding particle
//		  * coll_data[1] keeps id of colliding body
//		  */
//		glm::vec3 position = positions[coll_data[0]];
//		glm::vec3 projection = positions[coll_data[0]];
//		glm::float_t mass = masses_inv[coll_data[0]];
//		glm::vec3 force = forces[coll_data[0]];
//		glm::vec3 velocity = velocities[coll_data[0]];
//
//		CollisionBodyData body = collisions_bodies_data[coll_data[1]];
//
//		if (body.type == CollisionBody.TRIANGLE) {
//			glm::vec3 tri0 = positions[body.data.triangle.idx[0]];
//			glm::vec3 tri1 = positions[body.data.triangle.idx[1]];
//			glm::vec3 tri2 = positions[body.data.triangle.idx[2]];
//
//			// for barycentric coord test
//			glm::vec3 a = tri1 - tri0;
//			glm::vec3 b = tri2 - tri0;
//
//			glm::vec3 norm = glm::cross(a, b);
//			glm::vec3 diff = projection - position;
//			glm::float_t k = dot(norm, diff);
//			if (k < 0.0001f)
//				return;
//
//			k = dot(norm, (tri1 - position)) / k;
//			projection = position + k * diff;
//
//			// barycentric coord test
//			glm::vec3 c = projection - tri0;
//
//			glm::float_t dot00 = dot(a, a);
//			glm::float_t dot01 = dot(a, b);
//			glm::float_t dot02 = dot(a, c);
//			glm::float_t dot11 = dot(b, b);
//			glm::float_t dot12 = dot(b, c);
//
//			glm::float_t den = 1 / (dot00 * dot11 - dot01 * dot01);
//			glm::float_t u = (dot11 * dot02 - dot01 * dot12) * den;
//			glm::float_t v = (dot00 * dot12 - dot01 * dot02) * den;
//
//			if (u < 0 || v < 0 || u + v >= 1)
//				return;
//		}
//
//		projections[coll_data[0]] = projection;
//		velocities[coll_data[0]] = velocity;
//		forces[coll_data[0]] = force;
//	}
//}
//
///**
//  */
//
///**
//  step 6. Integrate motion.
//  */
//__global__ void integrateMotion(
//		glm::float_t dt,
//		glm::vec3 *positions,
//		glm::vec3 *projections,
//		glm::vec3 *velocities,
//		glm::uint max_idx
//		)
//{
//	int idx = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (idx < max_idx) {
//		glm::vec3 pos = positions[idx];
//		glm::vec3 proj = projections[idx];
//
//		velocities[idx] = (proj - pos) * dt;
//		positions[idx] = proj
//	}
//}

struct BufferMapping {
	glm::vec3 *vboPtr;
	unsigned int baseIdx;
};

__global__ void cudaUpdateVertexBuffers(BufferMapping mapp, glm::vec3 *positions, glm::uint *mapping, glm::uint max_idx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		glm::uint index = mapping[idx];
		glm::vec3 vertex = positions[index];
		vertex -= glm::vec3(0, 0.1, 0);
		mapp.vboPtr[idx - mapp.baseIdx] = vertex;
	}
}

void CUDASoftBodySolver::cudaUpdateVertexBuffer(glm::vec3 *positions, glm::uint
		*mapping, glm::vec3 *vboPtr, unsigned int baseIdx,
		unsigned int len)
{
	BufferMapping b;
	b.vboPtr = vboPtr;
	b.baseIdx = baseIdx;
	cudaUpdateVertexBuffers<<<10, 512>>>(b, positions, mapping, len);
	ERR("Kernel run");
}
