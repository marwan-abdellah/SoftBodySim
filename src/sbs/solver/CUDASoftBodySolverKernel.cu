#include "hip/hip_runtime.h"
#include "common.h"

#define GL_FORCE_CUDA
#include <glm/glm.hpp>
#include "sbs/solver/Math.h"
#include "sbs/solver/CUDASoftBodySolverKernel.h"

__global__ void cudaProjectPositionsAndVelocitiesKernel(
		glm::vec3 gravity,
		glm::vec3 *positions,
		glm::vec3 *projections,
		glm::vec3 *velocities,
		glm::vec3 *ext_forces,
		glm::float_t *masses,
		glm::float_t dt,
		glm::uint_t max_idx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx < max_idx) {

		// 0. Load from global mem.
		glm::vec3 force(0,0,0);
		if (ext_forces)
			glm::vec3 force = ext_forces[idx];

		glm::vec3 position = positions[idx];
		glm::float_t imass = masses[idx];
		glm::vec3 velocity = velocities[idx];
		
		// 1. Updating velocities.
		velocity += dt * imass * (force + gravity);

		// 2. Damp velocities.
		velocity *= 0.99f; // Naive damping

		// 3. projecting positions
		glm::vec3 projection = position + velocity * dt;

		// update global tables
		projections[idx] = projection;
		velocities[idx] = velocity;
	}
}

__global__ void solveShapeMatchingConstraints1(
		ParticleInfo *info_array,
		ShapeRegionInfo *regions,
		glm::uint_t *members_offsets,
		glm::vec3 *shapes_init_positions,
		glm::vec3 *projections,
		glm::float_t *masses,
		glm::uint_t max_idx
		)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		glm::mat3 A(0);
		glm::vec3 mc(0,0,0);
		glm::mat3 R, S;
		ParticleInfo info = info_array[idx];
		ShapeRegionInfo reg = regions[info.region_id];

		for (int i = 0; i < reg.n_particles; ++i) {
			glm::uint_t *members = members_offsets +
				reg.members_offsets_offset;
			glm::uint mem_offset = members[i];

			glm::vec3 *inits = shapes_init_positions +
				reg.shapes_init_positions_offset;
			glm::vec3 init = inits[mem_offset];

			glm::uint_t offset = info.body_offset + mem_offset;
			glm::vec3 proj = projections[offset];
			glm::float_t mass = masses[offset];
			A += glm::outerProduct(proj, init);
			mc += proj * mass;
		}

		mc = mc / reg.mass;
		A -= reg.mass * glm::outerProduct(mc, reg.mc0);

		polar_decomposition(A, R, S);

		for (int i = 0; i < reg.n_particles; ++i) {
			glm::uint_t *members = members_offsets +
				reg.members_offsets_offset;
			glm::uint mem_offset = members[i];

			glm::vec3 *inits = shapes_init_positions +
				reg.shapes_init_positions_offset;
			glm::vec3 init = inits[mem_offset];

			glm::vec3 final = R * (init - reg.mc0) + mc;

			atomicAdd(&projections[info.body_offset + mem_offset][0], final[0]);
			atomicAdd(&projections[info.body_offset + mem_offset][1], final[1]);
			atomicAdd(&projections[info.body_offset + mem_offset][2], final[2]);
		}
	}
}

#if 0
__global__ void solveShapeMatchingConstraints2(
		ParticleInfo *info,
		glm::vec3 *projections,
		glm::uvec3 *triangles,
		glm::uint_t max_idx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		ParticleInfo info = info[idx];
		glm::vec3 v0 = projections[info.body_offset + triangles[i][0]];
		glm::vec3 v1 = projections[info.body_offset + triangles[i][1]];
		glm::vec3 v2 = projections[info.body_offset + triangles[i][2]];
	}
}
#endif

__global__ void solveShapeMatchingConstraints2(
		ParticleInfo *info_array,
		ShapeRegionInfo *regions,
		glm::vec3 *projections,
		glm::uint_t max_idx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		ParticleInfo info = info_array[idx];
		ShapeRegionInfo reg = regions[info.region_id];
		glm::vec3 proj = projections[idx];

		proj = proj / (glm::float_t)(reg.n_regions + 1);

		projections[idx] = proj;
	}
}

__global__ void solveGroundWallCollisionConstraints(
		glm::vec3 *projections,
		glm::float_t *masses_inv,
		glm::float_t ground_level,
		glm::float_t left_wall,
		glm::float_t right_wall,
		glm::float_t front_wall,
		glm::float_t back_wall,
		glm::uint_t max_idx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		glm::vec3 pos = projections[idx];

		pos[1] = pos[1] < ground_level ? ground_level : pos[1];
		pos[0] = pos[0] < left_wall ? left_wall : pos[0];
		pos[0] = pos[0] > right_wall ? right_wall : pos[0];
		pos[2] = pos[2] > front_wall ? front_wall : pos[2];
		pos[2] = pos[2] < back_wall ? back_wall : pos[2];

		projections[idx] = pos;
	}
}

/**
  step 6. Integrate motion.
  */
__global__ void integrateMotionKernel(
		glm::float_t dt,
		glm::vec3 *positions,
		glm::vec3 *projections,
		glm::vec3 *velocities,
		glm::uint max_idx
		)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		glm::vec3 pos = positions[idx];
		glm::vec3 proj = projections[idx];

		velocities[idx] = (proj - pos) / dt;
		positions[idx] = proj;
	}
}

__global__ void cudaUpdateVertexBufferKernel(glm::vec3 *vboPtr, glm::vec3
		*positions, glm::uint *mapping, glm::uint_t baseIdx, glm::uint_t mappingBaseIdx,
		glm::uint max_idx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		glm::uint index = mapping[mappingBaseIdx + idx];
		glm::vec3 vertex = positions[baseIdx + index];
		vboPtr[idx] = vertex;
	}
}

#if 0
__global__ void calculateLinkStiffness(
		unsigned int solver_steps,
		LinkConstraint *links,
		glm::uint_t linkBaseIdx,
		glm::uint_t max_idx)
{
	int link_idx = blockIdx.x * blockDim.x + threadIdx.x + linkBaseIdx;

	if (link_idx < max_idx) {
		LinkConstraint lnk = links[link_idx];

		links[link_idx].stiffness = 1.0f - powf(1.0 - lnk.stiffness, 1.0f /
				solver_steps);
	}
}

__device__ glm::uint_t hash(glm::uint_t id)
{
	return 1193 * id;
}


/**
  step 4. solving links constraints.
  */
__global__ void solveLinksConstraints(
		unsigned int max_steps,
		LinkConstraint *links,
		glm::vec3 *projections,
		glm::float_t *masses_inv,
		glm::uint_t baseIdx,
		glm::uint_t linkIdx,
		glm::uint_t max_idx)
{
	__shared__ glm::vec3   ACCUM[2 * MAX_LINKS];
	__shared__ glm::uint_t COUNTER[2 * MAX_LINKS];

	int link_idx = blockIdx.x * blockDim.x + threadIdx.x + linkIdx;

	if (link_idx < max_idx) {

		LinkConstraint lnk = links[link_idx];
		glm::vec3 pos0 = projections[lnk.index[0] + baseIdx];
		glm::vec3 pos1 = projections[lnk.index[1] + baseIdx];
		glm::float_t mass_inv0 = masses_inv[lnk.index[0] + baseIdx];
		glm::float_t mass_inv1 = masses_inv[lnk.index[1] + baseIdx];

		// assume that will be no colliosions; MAX_LINS = 2^x, X in N
		glm::uint_t id0 = hash(lnk.index[0]) & (2 * MAX_LINKS - 1);
		glm::uint_t id1 = hash(lnk.index[1]) & (2 * MAX_LINKS - 1);

		ACCUM[id0] = pos0;
		ACCUM[id1] = pos1;
		COUNTER[id0] = 1;
		COUNTER[id1] = 1;

		__syncthreads();

		glm::float_t restLen = lnk.restLength;
		glm::float_t k = lnk.stiffness;

		glm::vec3 diff = pos0 - pos1;
		glm::float_t len = glm::length(diff);

		glm::float_t m0 = mass_inv0 / (mass_inv0 + mass_inv1) * (len - restLen) /
			len;
		glm::float_t m1 = mass_inv1 / (mass_inv0 + mass_inv1) * (len - restLen) /
			len;

		pos0 -= k * m0 * diff;
		pos1 += k * m1 * diff;

		atomicAdd(&ACCUM[id0][0], pos0[0]);
		atomicAdd(&ACCUM[id0][1], pos0[1]);
		atomicAdd(&ACCUM[id0][2], pos0[2]);
		atomicAdd(&ACCUM[id1][0], pos1[0]);
		atomicAdd(&ACCUM[id1][1], pos1[1]);
		atomicAdd(&ACCUM[id1][2], pos1[2]);

		atomicInc(&COUNTER[id0], MAX_LINKS);
		atomicInc(&COUNTER[id1], MAX_LINKS);

		__syncthreads();

		pos0 = ACCUM[id0] * (1.0f / (glm::float_t)COUNTER[id0]);
		pos1 = ACCUM[id1] * (1.0f / (glm::float_t)COUNTER[id1]);

		projections[lnk.index[0] + baseIdx] = pos0;
		projections[lnk.index[1] + baseIdx] = pos1;
	}
}
#endif
