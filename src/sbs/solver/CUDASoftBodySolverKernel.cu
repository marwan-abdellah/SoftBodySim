#include "hip/hip_runtime.h"
#include "common.h"

#include <glm/glm.hpp>
#include "sbs/solver/Math.h"
#include "sbs/solver/CUDASoftBodySolverKernel.h"

__device__ glm::uint_t hash(glm::uint_t id)
{
	return 1193 * id;
}

__global__ void cudaProjectPositionsAndVelocitiesKernel(
		glm::vec3 gravity,
		glm::vec3 *positions,
		glm::vec3 *projections,
		glm::vec3 *velocities,
		glm::vec3 *ext_forces,
		glm::float_t *masses,
		glm::float_t dt,
		glm::uint_t max_idx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( idx < max_idx) {

		// 0. Load from global mem.
		glm::vec3 force(0,0,0);
		if (ext_forces)
			glm::vec3 force = ext_forces[idx];

		glm::vec3 position = positions[idx];
		glm::float_t imass = masses[idx];
		glm::vec3 velocity = velocities[idx];
		
		// 1. Updating velocities.
		velocity += dt * imass * (force + gravity);

		// 2. Damp velocities.
		velocity *= 0.99f; // Naive damping

		// 3. projecting positions
		glm::vec3 projection = position + velocity * dt;

		// update global tables
		projections[idx] = projection;
		velocities[idx] = velocity;
	}
}

/**
  step 4. solving links constraints.
  */
__global__ void solveLinksConstraints(
		unsigned int max_steps,
		LinkConstraint *links,
		glm::vec3 *projections,
		glm::float_t *masses_inv,
		glm::uint_t baseIdx,
		glm::uint_t linkIdx,
		glm::uint_t max_idx)
{
	__shared__ glm::vec3   ACCUM[2 * MAX_LINKS];
	__shared__ glm::uint_t COUNTER[2 * MAX_LINKS];

	int link_idx = blockIdx.x * blockDim.x + threadIdx.x + linkIdx;

	if (link_idx < max_idx) {

		LinkConstraint lnk = links[link_idx];
		glm::vec3 pos0 = projections[lnk.index[0] + baseIdx];
		glm::vec3 pos1 = projections[lnk.index[1] + baseIdx];
		glm::float_t mass_inv0 = masses_inv[lnk.index[0] + baseIdx];
		glm::float_t mass_inv1 = masses_inv[lnk.index[1] + baseIdx];

		// assume that will be no colliosions; MAX_LINS = 2^x, X in N
		glm::uint_t id0 = hash(lnk.index[0]) & (2 * MAX_LINKS - 1);
		glm::uint_t id1 = hash(lnk.index[1]) & (2 * MAX_LINKS - 1);

		ACCUM[id0] = pos0;
		ACCUM[id1] = pos1;
		COUNTER[id0] = 1;
		COUNTER[id1] = 1;

		__syncthreads();

		glm::float_t restLen = lnk.restLength;
		glm::float_t k = lnk.stiffness;

		glm::vec3 diff = pos0 - pos1;
		glm::float_t len = glm::length(diff);

		glm::float_t m0 = mass_inv0 / (mass_inv0 + mass_inv1) * (len - restLen) /
			len;
		glm::float_t m1 = mass_inv1 / (mass_inv0 + mass_inv1) * (len - restLen) /
			len;

		pos0 -= k * m0 * diff;
		pos1 += k * m1 * diff;

		atomicAdd(&ACCUM[id0][0], pos0[0]);
		atomicAdd(&ACCUM[id0][1], pos0[1]);
		atomicAdd(&ACCUM[id0][2], pos0[2]);
		atomicAdd(&ACCUM[id1][0], pos1[0]);
		atomicAdd(&ACCUM[id1][1], pos1[1]);
		atomicAdd(&ACCUM[id1][2], pos1[2]);

		atomicInc(&COUNTER[id0], MAX_LINKS);
		atomicInc(&COUNTER[id1], MAX_LINKS);

		__syncthreads();

		pos0 = ACCUM[id0] * (1.0f / (glm::float_t)COUNTER[id0]);
		pos1 = ACCUM[id1] * (1.0f / (glm::float_t)COUNTER[id1]);

		projections[lnk.index[0] + baseIdx] = pos0;
		projections[lnk.index[1] + baseIdx] = pos1;
	}
}

__global__ void solveGroundCollisionConstraints(
		glm::vec3 *projections,
		glm::float_t *masses_inv,
		glm::float_t ground_level,
		glm::uint_t max_idx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		glm::vec3 pos = projections[idx];

		if (pos[1] < ground_level)
			pos[1] = ground_level;

		projections[idx] = pos;
	}
}

/**
  step 6. Integrate motion.
  */
__global__ void integrateMotionKernel(
		glm::float_t dt,
		glm::vec3 *positions,
		glm::vec3 *projections,
		glm::vec3 *velocities,
		glm::uint max_idx
		)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		glm::vec3 pos = positions[idx];
		glm::vec3 proj = projections[idx];

		velocities[idx] = (proj - pos) / dt;
		positions[idx] = proj;
	}
}

__global__ void cudaUpdateVertexBufferKernel(glm::vec3 *vboPtr, glm::vec3
		*positions, glm::uint *mapping, glm::uint_t baseIdx, glm::uint_t mappingBaseIdx,
		glm::uint max_idx)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < max_idx) {
		glm::uint index = mapping[mappingBaseIdx + idx];
		glm::vec3 vertex = positions[baseIdx + index];
		vboPtr[idx] = vertex;
	}
}

__global__ void calculateLinkStiffness(
		unsigned int solver_steps,
		LinkConstraint *links,
		glm::uint_t linkBaseIdx,
		glm::uint_t max_idx)
{
	int link_idx = blockIdx.x * blockDim.x + threadIdx.x + linkBaseIdx;

	if (link_idx < max_idx) {
		LinkConstraint lnk = links[link_idx];

		links[link_idx].stiffness = 1.0f - powf(1.0 - lnk.stiffness, 1.0f /
				solver_steps);
	}
}
