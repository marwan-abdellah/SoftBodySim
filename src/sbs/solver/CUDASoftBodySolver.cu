#include "common.h"

#include "sbs/solver/CUDASoftBodySolver.h"
#include "sbs/solver/CUDAVector.h"
#include "sbs/solver/CUDASoftBodySolverKernel.h"

#include <cstring>

using namespace std;

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define DEFAULT_SOLVER_STEPS 10
#define DEFAULT_CELL_SIZE 1.0
#define MAX_REGION_SIZE 10

struct ShapeRegion {
	glm::vec3 mc0; // initial region mass center
	float_t mass; // region total mass
	glm::uint_t indexes[MAX_REGION_SIZE]; // indexes of region particles
};

struct ShapeDescriptor {
	glm::vec3 mc0; // body global mass
	CUDAVector<glm::vec3> initPos; // initial particle locations (x0i);
	float_t radius; // maximum distance between mass center and particle;
	float_t massTotal; // object total mass
	float_t volume; // initial shape volume
	CUDAVector<ShapeRegion> regions; // shape regions
};

class HIPContext {
public:
	typedef std::vector<SoftBodyDescriptor> descriptorArray_t;

	//cudaContextCreate(softbodyList_t*);
	HIPContext(softbodyList_t *list);

	//void cudaContextShutdown(SolverPrivate*);
	~HIPContext(void);

	bool InitDevice();
	bool ShutdownDevice();

	void UpdateVertexBuffers(bool async);
	void ProjectSystem(glm::float_t dt, CUDASoftBodySolver::SoftBodyWorldParameters
			&parms);
	bool InitSoftBody(SoftBody *body);
private:
	void UpdateConstraintStiffness(SoftBodyDescriptor &descr, int mSolverSteps);
	SoftBodyDescriptor CreateDescriptor(SoftBody *body);
	bool RegisterVertexBuffers(SoftBodyDescriptor &descr);
	hipGraphicsResource *RegisterGLGraphicsResource(const VertexBuffer *vb);

	int                                mDeviceId;
	hipDeviceProp_t                     mDevProp;
	hipStream_t                       mStream;
	int                                mSolverSteps;

	descriptorArray_t                  mDescriptors;
	CUDAVector<SoftBodyDescriptor>	   mDescriptorsDev;
	CUDAVector<ShapeDescriptor>        mShapeDescriptors;

	CUDAVector<glm::vec3>              mPositions;
	CUDAVector<glm::vec3>              mProjections;
	CUDAVector<glm::vec3>              mVelocities;
	CUDAVector<glm::float_t>           mInvMasses;
	CUDAVector<glm::vec3>              mForces;

	CUDAVector<LinkConstraint>         mLinks;
	CUDAVector<glm::uint_t>            mMapping;
	CUDAVector<glm::uvec3>             mTriangles;

	vector<hipGraphicsResource*>      mResArray; /* helper array to map all resources 
												  in one call */
};

bool HIPContext::InitDevice()
{
	hipError_t err;
	hipDeviceProp_t  prop;
	memset(&prop, 0x0, sizeof(prop));
	prop.major = 3;
	prop.minor = 5;

	// choose device for us. Prefer with compute capabilities ~ 3.5
	err = hipChooseDevice(&mDeviceId, &prop);
	if (err != hipSuccess) goto on_error;

	err = hipSetDevice(mDeviceId);
	if (err != hipSuccess) goto on_error;

	err = hipGetDeviceProperties(&mDevProp, mDeviceId);
	if (err != hipSuccess) goto on_error;
	
	err = hipStreamCreate(&mStream);
	if (err != hipSuccess) goto on_error;

	DBG("Choosen CUDA Device: %s", mDevProp.name);

	return true;

on_error:
	ERR("Device initialization error: %s", hipGetErrorString(hipGetLastError()));
	return false;
}

bool HIPContext::ShutdownDevice()
{
	hipError_t err;

	err = hipDeviceSynchronize();
	if (err != hipSuccess) return false;

	err = hipDeviceReset();
	if (err != hipSuccess) return false;


	return true;
}

SoftBodyDescriptor HIPContext::CreateDescriptor(SoftBody *body)
{
	SoftBodyDescriptor descr;

	descr.body = body;
	descr.graphics = NULL;
	descr.baseIdx = mPositions.size();
	descr.nParticles = body->mParticles.size();
	descr.linkIdx = mLinks.size();
	descr.nLinks = body->mLinks.size();
	descr.mappingIdx = mMapping.size();
	descr.nMapping = body->mMeshVertexParticleMapping.size();
	descr.trianglesIdx = mTriangles.size();
	descr.nTriangles = body->mTriangles.size();

	return descr;
}

hipGraphicsResource *HIPContext::RegisterGLGraphicsResource(const VertexBuffer *vb)
{
	hipError_t err;
	hipGraphicsResource *ret = NULL;
	GLuint id = vb->GetVBO();
	err = hipGraphicsGLRegisterBuffer(&ret, id, hipGraphicsRegisterFlagsNone);
	if (err != hipSuccess) {
		ERR("Unable to register GL buffer object %d", id);
		return NULL;
	}
	return ret;
}

bool HIPContext::RegisterVertexBuffers(SoftBodyDescriptor &descr)
{
	if (!descr.body) {
		ERR("No SoftBody reference in descriptor!");
		return false;
	}

	const VertexBuffer *buf = descr.body->GetVertexes();
	if (buf)
		descr.graphics = RegisterGLGraphicsResource(buf);

	return true;
}

void HIPContext::UpdateConstraintStiffness(SoftBodyDescriptor
		&descr, int mSolverSteps)
{
	int blocks = descr.nLinks / 128;
	calculateLinkStiffness<<<blocks, 128>>>(mSolverSteps, mLinks.data(),
			descr.linkIdx, descr.nLinks);
}

bool HIPContext::InitSoftBody(SoftBody *body)
{
	SoftBodyDescriptor descr = CreateDescriptor(body);
	bool res;

	mPositions.push_back(&(body->mParticles[0]), descr.nParticles);
	mProjections.push_back(&(body->mParticles[0]), descr.nParticles);
	mVelocities.resize(mVelocities.size() + descr.nParticles);
	mInvMasses.push_back(&(body->mMassInv[0]), descr.nParticles);
	mForces.resize(mForces.size() + descr.nParticles);
	mLinks.push_back(&(body->mLinks[0]), descr.nLinks);
	mMapping.push_back(&(body->mMeshVertexParticleMapping[0]), descr.nMapping);

	res = RegisterVertexBuffers(descr);
	if (!res) {
		ERR("Error occured registering SoftBody vertex buffers.");
		ERR("Cuda error: %s", hipGetErrorString(hipGetLastError()));
		return false;
	}

	UpdateConstraintStiffness(descr, mSolverSteps);

	mDescriptors.push_back(descr);
	mDescriptorsDev.push_back(descr);
	mResArray.push_back(descr.graphics);

	return true;
}

#if 0
bool HIPContext::InitDymmyBodyCollisionConstraint()
{
	long int total = 0, bytes = 0;
	vector<PointTriangleConstraint> constraints;
	PointTriangleConstraint con;

	// constant collision handling
	// create m * x collsion constraints - to be optimized later.
	FOREACH(it, &mDescriptors) {
		constraints.clear();
		FOREACH(vx, &it->body->mPositions) {
			int idx = std::distance(it->body->mPositions.begin(), vx);
			con.pointObjectId = std::distance(mDescriptors.begin(), it);
			con.pointIdx = idx;
			/*
			FOREACH(tr, &it->body->mTriangles) {
				if (idx == (*tr)[0] ||
					idx == (*tr)[1] ||
					idx == (*tr)[2]) continue;
				con.triangleObjectId = std::distance(mDescriptors.begin(), it);
				con.triangleId = std::distance(it->body->mTriangles.begin(), tr);
				constraints.push_back(con);
			}
			*/
			FOREACH(it2, &mDescriptors) {
				if (it == it2) continue;
				FOREACH(tr, &it->body->mTriangles) {
					con.triangleObjectId = std::distance(
							mDescriptors.begin(), it2);
					con.triangleId = std::distance(it->body->mTriangles.begin(),
							tr);
					constraints.push_back(con);
					total++;
				}
			}
		}
		if (it->collisions) hipFree(it->collisions);
		it->collisions = (PointTriangleConstraint*)allocateCUDABuffer(sizeof(PointTriangleConstraint) * constraints.size());
		it->nCollisions = constraints.size();
		hipMemcpy(it->collisions, &constraints[0],
				sizeof(PointTriangleConstraint) * constraints.size(),
				hipMemcpyHostToDevice);
		bytes += sizeof(PointTriangleConstraint) * constraints.size();
	}
	DBG("allocated constraints %d, bytes %d", total, bytes);
	return true;
}
#endif

HIPContext::HIPContext(softbodyList_t *bodies)
{
	mSolverSteps = DEFAULT_SOLVER_STEPS;

	if (!InitDevice()) {
		ERR("CUDA Device initialization failed!");
		return;
	}

	FOREACH(it, bodies) {
		if (!*it) continue;

		if (!InitSoftBody(*it)) {
			ShutdownDevice();
			return;
		}
	}

#if 0
	if (!InitDymmyBodyCollisionConstraint()) {
		ERR("Unable to allocate collision constraints on device!");
		ShutdownDevice();
		return;
	}
#endif 
}

HIPContext::~HIPContext()
{
	ShutdownDevice();
}

void HIPContext::UpdateVertexBuffers(bool async)
{
	hipError_t err;
	glm::vec3 *ptr;
	int threadsPerBlock = 128;

	// map all in one call
	err = hipGraphicsMapResources(mResArray.size(), &mResArray[0]);
	if (err != hipSuccess) return;

	FOREACH(it, &mDescriptors) {
		size_t size;
		err = hipGraphicsResourceGetMappedPointer((void**)&ptr, &size, it->graphics);
		if (err != hipSuccess) {
			ERR("Unable to map VBO pointer");
			return;
		}
		int blockCount = it->nMapping / threadsPerBlock + 1;
		cudaUpdateVertexBufferKernel<<<blockCount, threadsPerBlock >>>(
				ptr, mPositions.data(), mMapping.data(), it->baseIdx, 
				it->mappingIdx, it->nMapping);
	}

	hipGraphicsUnmapResources(mResArray.size(), &mResArray[0]);
}

CUDASoftBodySolver::CUDASoftBodySolver(void)
	:
		mContext(0),
		mInitialized(false)
{
}

CUDASoftBodySolver::~CUDASoftBodySolver(void)
{
	if (mContext) delete mContext;
}

bool CUDASoftBodySolver::Initialize(void)
{
	if (mInitialized) return true;

	mContext = new HIPContext(&mBodies);
		if (!mContext) {
		ERR("Unable to create CUDA context.");
		return false;
	}

	mInitialized = true;
	return true;
}

void CUDASoftBodySolver::Shutdown(void)
{
	if (!mInitialized) return;

	if (mContext) {
		delete mContext;
		mContext = NULL;
	}
	mInitialized = false;
	SoftBodySolver::Shutdown();
}

void CUDASoftBodySolver::UpdateVertexBuffers(void)
{
	if (mInitialized)
		mContext->UpdateVertexBuffers(false);
}

void HIPContext::ProjectSystem(glm::float_t dt, CUDASoftBodySolver::SoftBodyWorldParameters &world)
{
	int threadsPerBlock = 128;
	int blockCount;

	if (!mPositions.size()) return;

	// predict motion
	blockCount = mPositions.size() / threadsPerBlock + 1;
	cudaProjectPositionsAndVelocitiesKernel<<<blockCount,
		threadsPerBlock>>>(world.gravity, mPositions.data(),
			mProjections.data(), mVelocities.data(), NULL,
			mInvMasses.data(), dt, mPositions.size());

	// solver
	blockCount = mPositions.size() / threadsPerBlock + 1;
	FOREACH(it, &mDescriptors) {
		int linkBlockCount = it->nLinks / MAX_LINKS + 1;

		solveLinksConstraints<<<linkBlockCount, threadsPerBlock>>>(
				1, mLinks.data(), mProjections.data(), mInvMasses.data(),
				it->baseIdx, it->linkIdx, it->nLinks);
	}
	solveGroundWallCollisionConstraints<<<blockCount, threadsPerBlock>>>(
			mProjections.data(), mInvMasses.data(),
			world.groundLevel, world.leftWall, world.rightWall,
			world.frontWall, world.backWall, mPositions.size());

	// integrate motion
	threadsPerBlock = 128;
	blockCount = mPositions.size() / threadsPerBlock + 1;
	integrateMotionKernel<<<blockCount, threadsPerBlock>>>(
			dt, mPositions.data(), mProjections.data(),
			mVelocities.data(), mPositions.size());
}

void CUDASoftBodySolver::ProjectSystem(glm::float_t dt)
{
	if (mInitialized)
		mContext->ProjectSystem(dt, mWorldParams);
}

void CUDASoftBodySolver::AddSoftBody(SoftBody *body)
{
	mBodies.push_back(body);
	if (!mInitialized || !mContext->InitSoftBody(body))
		ERR("Failed to add SoftBody!");
}
