#include "hip/hip_runtime.h"
#include "common.h"

#include "sbs/solver/CUDASoftBodySolver.h"
#include "sbs/solver/CUDAVector.h"
#include "sbs/solver/Math.h"
#include "sbs/solver/CUDASoftBodySolverKernel.h"

#include <cstring>
#include <set>
#include <queue>

using namespace std;

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#define DEFAULT_SOLVER_STEPS 10

class HIPContext {
public:
	typedef std::vector<SoftBodyDescriptor> descriptorArray_t;

	//cudaContextCreate(softbodyList_t*);
	HIPContext(softbodyList_t *list);

	//void cudaContextShutdown(SolverPrivate*);
	~HIPContext(void);

	bool InitDevice();
	bool ShutdownDevice();

	void UpdateVertexBuffers(bool async);
	void ProjectSystem(glm::float_t dt, CUDASoftBodySolver::SoftBodyWorldParameters
			&parms);
	bool InitSoftBody(SoftBody *body);
private:
	void UpdateConstraintStiffness(SoftBodyDescriptor &descr, int mSolverSteps);
	void CreateDescriptor(SoftBody *body);
	void CreateShapeDescriptor(SoftBody *body);
	bool RegisterVertexBuffers(SoftBodyDescriptor &descr);
	hipGraphicsResource *RegisterGLGraphicsResource(const VertexBuffer *vb);

	int                                mDeviceId;
	hipDeviceProp_t                     mDevProp;
	hipStream_t                       mStream;
	int                                mSolverSteps;

	descriptorArray_t                  mDescriptors;
	CUDAVector<SoftBodyDescriptor>	   mDescriptorsDev;
	CUDAVector<ShapeDescriptor>        mShapeDescriptors;

	// shape matching
	CUDAVector<ShapeRegionStaticInfo>  mRegions;
	CUDAVector<ShapeRegionDynamicInfo> mRegionsDynamicInfo;
	CUDAVector<glm::uint_t>            mRegionsMembersOffsets;
	CUDAVector<glm::uint_t>            mMembersRegionsOffsets;
	CUDAVector<glm::vec3>              mShapeInitialPositions; // initial particle locations (x0i);
	CUDAVector<glm::float_t>           mPartials;

	CUDAVector<ParticleInfo>           mParticlesInfo;
	CUDAVector<glm::vec3>              mPositions;
	CUDAVector<glm::vec3>              mProjections;
	CUDAVector<glm::vec3>              mVelocities;
	CUDAVector<glm::float_t>           mInvMasses;
	CUDAVector<glm::vec3>              mForces;

	CUDAVector<LinkConstraint>         mLinks;
	CUDAVector<glm::uint_t>            mMapping;
	CUDAVector<glm::uvec3>             mTriangles;
	CUDAVector<glm::vec3>              mTrianglesNormals;
	CUDAVector<ParticleTrianglesInfo>  mParticleTriangleInfo;
	CUDAVector<glm::uint_t>            mParticleTriangleIndexes;

	vector<hipGraphicsResource*>      mResArray; /* helper array to map all resources 
												  in one call */
};

bool HIPContext::InitDevice()
{
	hipError_t err;
	hipDeviceProp_t  prop;
	memset(&prop, 0x0, sizeof(prop));
	prop.major = 3;
	prop.minor = 5;

	// choose device for us. Prefer with compute capabilities ~ 3.5
	err = hipChooseDevice(&mDeviceId, &prop);
	if (err != hipSuccess) goto on_error;

	err = hipSetDevice(mDeviceId);
	if (err != hipSuccess) goto on_error;

	err = hipGetDeviceProperties(&mDevProp, mDeviceId);
	if (err != hipSuccess) goto on_error;
	
	err = hipStreamCreate(&mStream);
	if (err != hipSuccess) goto on_error;

	DBG("Choosen CUDA Device: %s", mDevProp.name);

	return true;

on_error:
	ERR("Device initialization error: %s", hipGetErrorString(hipGetLastError()));
	return false;
}

bool HIPContext::ShutdownDevice()
{
	hipError_t err;

	err = hipDeviceSynchronize();
	if (err != hipSuccess) return false;

	err = hipDeviceReset();
	if (err != hipSuccess) return false;

	return true;
}

struct Node {
	Node(int i, int d) : idx(i), distance(d) {}
	int idx;
	int distance;
};

void GetRegion(int idx, const MeshData::neighboursArray_t &nei, int max, indexArray_t &out)
{
	std::queue<Node> toprocess;
	std::set<int> processed;

	toprocess.push(Node(idx, 0));

	while (!toprocess.empty()) {
		Node n = toprocess.front();
		if (processed.find(n.idx) == processed.end()) {
			out.push_back(n.idx);
			processed.insert(n.idx);
		}
		toprocess.pop();

		if (n.distance >= max) continue;

		FOREACH_R(it, nei[n.idx])
			toprocess.push(Node(*it, n.distance + 1));
	}
}

void HIPContext::CreateShapeDescriptor(SoftBody *obj)
{
	ShapeDescriptor d;
	vec3Array_t initQ;
	long len = 0;
	unsigned int smin = 999999;
	unsigned int smax = 0;
	int region_size = 2;

	d.mc0 = calculateMassCenter(
			&(obj->mParticles[0]), &(obj->mMassInv[0]), obj->mParticles.size());

	d.initPosBaseIdx = mShapeInitialPositions.size(); // shuld depend on mesh
	mShapeInitialPositions.push_back(&(obj->mParticles[0]), obj->mParticles.size());

	d.radius = 0;
	const MeshData::neighboursArray_t &na = obj->mMesh->GetNeighboursArray();

	int regions_base_id = mRegions.size();

	std::vector< std::vector<glm::uint_t> > particlesInRegions;
	particlesInRegions.resize(obj->mParticles.size());

	REP(i, obj->mParticles.size()) {
		indexArray_t indexes;
		GetRegion(i, na, region_size, indexes);
		REP(p, indexes.size()) {
			particlesInRegions[indexes[p]].push_back(i);
		}
	}
	// triangles info
	std::vector< std::set<glm::uint_t> > particlesInTriangles;
	std::vector< std::vector<glm::uint_t> > particlesInTriangles2;
	particlesInTriangles.resize(obj->mParticles.size());
	particlesInTriangles2.resize(obj->mParticles.size());
	REP(i, obj->mTriangles.size()) {
		glm::uvec3 idxs = obj->mTriangles[i];
		particlesInTriangles[idxs[0]].insert(i);
		particlesInTriangles[idxs[1]].insert(i);
		particlesInTriangles[idxs[2]].insert(i);
	}
	REP(i, particlesInTriangles.size()) {
		FOREACH_R(it, particlesInTriangles[i])
			particlesInTriangles2[i].push_back(*it);
	}

#if 0
	eEP(i, particlesInRegions.size()) {
		printf("%d:", particlesInRegions[i].size());
		REP(j, particlesInRegions[i].size()) {
			printf("%d,", particlesInRegions[i][j]);
		}
		printf("\n");
	}
#endif
	// create shape regions
	REP(i, obj->mParticles.size()) {
		ShapeRegionStaticInfo reg;
		ParticleInfo info;
		info.region_id = mRegions.size();
		info.body_info_id = mShapeDescriptors.size();
		info.body_offset = mPositions.size();
		indexArray_t indexes;
		float_t mass = 0.0f;
		glm::vec3 mc(0,0,0);

		GetRegion(i, na, region_size, indexes);

		len += indexes.size();
		if (smin > indexes.size())
			smin = indexes.size();
		if (smax < indexes.size())
			smax = indexes.size();

		FOREACH_R(it, indexes) {
			mass += obj->mMassInv[*it];
			mc += obj->mParticles[*it] * obj->mMassInv[*it];
		}
		reg.mass = mass;
		reg.mc0 = mc / mass;
		reg.n_particles = indexes.size();
		reg.members_offsets_offset = mRegionsMembersOffsets.size();
		reg.shapes_init_positions_offset = d.initPosBaseIdx;
		reg.regions_offsets_offset = mMembersRegionsOffsets.size();
		reg.n_regions = particlesInRegions[i].size();

		ParticleTrianglesInfo pti;
		pti.triangle_id_offset = mParticleTriangleIndexes.size();
		pti.n_triangles = particlesInTriangles2[i].size();

		mRegions.push_back(reg);
		mRegionsMembersOffsets.push_back(&indexes[0], indexes.size());
		mParticlesInfo.push_back(info);
		mMembersRegionsOffsets.push_back(&(particlesInRegions[i][0]),
				particlesInRegions[i].size());
		mParticleTriangleInfo.push_back(pti); 
		mParticleTriangleIndexes.push_back(&(particlesInTriangles2[i][0]),
				particlesInTriangles2[i].size());
	}

	DBG("==MODEL INFORMATION==");
	DBG("Particles total: %ld", obj->mParticles.size());
	DBG("Vertexes total: %ld", obj->mMesh->GetVertexes().size());
	DBG("Triangles total: %ld", obj->mMesh->GetFaces().size());
	DBG("Rest Volume :%f", d.volume);
	DBG("Regions total: %ld", mRegions.size());
	DBG("Average region size: %f", (float)len / mRegions.size());
	DBG("Max region size: %d", smax);
	DBG("Min region size: %d", smin);

	DBG("ParticeInfo size: %d", mParticlesInfo.size());

	mShapeDescriptors.push_back(d);
}

void HIPContext::CreateDescriptor(SoftBody *body)
{
	SoftBodyDescriptor descr;

	descr.body = body;
	descr.graphics = NULL;
	descr.baseIdx = mPositions.size();
	descr.nParticles = body->mParticles.size();
	descr.linkIdx = mLinks.size();
	descr.nLinks = body->mLinks.size();
	descr.mappingIdx = mMapping.size();
	descr.nMapping = body->mMeshVertexParticleMapping.size();
	descr.trianglesIdx = mTriangles.size();
	descr.nTriangles = body->mTriangles.size();

	descr.volume0 = calculateVolume(&(body->mParticles[0]),
			&(body->mTriangles[0]), NULL, NULL, body->mTriangles.size()); 

	bool res = RegisterVertexBuffers(descr);
	if (!res) {
		ERR("Error occured registering SoftBody vertex buffers.");
		return;
	}

	mDescriptors.push_back(descr);
	mDescriptorsDev.push_back(descr);
	mPartials.resize(body->mTriangles.size() / 128 + 1);
	mResArray.push_back(descr.graphics);
}

hipGraphicsResource *HIPContext::RegisterGLGraphicsResource(const VertexBuffer *vb)
{
	hipError_t err;
	hipGraphicsResource *ret = NULL;
	GLuint id = vb->GetVBO();
	err = hipGraphicsGLRegisterBuffer(&ret, id, hipGraphicsRegisterFlagsNone);
	if (err != hipSuccess) {
		ERR("Unable to register GL buffer object %d", id);
		return NULL;
	}
	return ret;
}

bool HIPContext::RegisterVertexBuffers(SoftBodyDescriptor &descr)
{
	if (!descr.body) {
		ERR("No SoftBody reference in descriptor!");
		return false;
	}

	const VertexBuffer *buf = descr.body->GetVertexes();
	if (buf)
		descr.graphics = RegisterGLGraphicsResource(buf);

	return true;
}

void HIPContext::UpdateConstraintStiffness(SoftBodyDescriptor
		&descr, int mSolverSteps)
{
	//int blocks = descr.nLinks / 128;
	//calculateLinkStiffness<<<blocks, 128>>>(mSolverSteps, mLinks.data(),
	//		descr.linkIdx, descr.nLinks);
}

bool HIPContext::InitSoftBody(SoftBody *body)
{
	CreateDescriptor(body);
	CreateShapeDescriptor(body);

	long nParticles = body->mParticles.size();
	mPositions.push_back(&(body->mParticles[0]), nParticles);
	mProjections.push_back(&(body->mParticles[0]), nParticles);
	mVelocities.resize(mVelocities.size() + nParticles);
	mInvMasses.push_back(&(body->mMassInv[0]), nParticles);
	mForces.resize(mForces.size() + nParticles);
	mLinks.push_back(&(body->mLinks[0]), body->mLinks.size());
	mMapping.push_back(&(body->mMeshVertexParticleMapping[0]),
			body->mMeshVertexParticleMapping.size());
	mRegionsDynamicInfo.resize(mPositions.size());
	mTriangles.push_back(&(body->mTriangles[0]), body->mTriangles.size());
	mTrianglesNormals.resize(body->mTriangles.size());

	return true;
}

#if 0
bool HIPContext::InitDymmyBodyCollisionConstraint()
{
	long int total = 0, bytes = 0;
	vector<PointTriangleConstraint> constraints;
	PointTriangleConstraint con;

	// constant collision handling
	// create m * x collsion constraints - to be optimized later.
	FOREACH(it, &mDescriptors) {
		constraints.clear();
		FOREACH(vx, &it->body->mPositions) {
			int idx = std::distance(it->body->mPositions.begin(), vx);
			con.pointObjectId = std::distance(mDescriptors.begin(), it);
			con.pointIdx = idx;
			/*
			FOREACH(tr, &it->body->mTriangles) {
				if (idx == (*tr)[0] ||
					idx == (*tr)[1] ||
					idx == (*tr)[2]) continue;
				con.triangleObjectId = std::distance(mDescriptors.begin(), it);
				con.triangleId = std::distance(it->body->mTriangles.begin(), tr);
				constraints.push_back(con);
			}
			*/
			FOREACH(it2, &mDescriptors) {
				if (it == it2) continue;
				FOREACH(tr, &it->body->mTriangles) {
					con.triangleObjectId = std::distance(
							mDescriptors.begin(), it2);
					con.triangleId = std::distance(it->body->mTriangles.begin(),
							tr);
					constraints.push_back(con);
					total++;
				}
			}
		}
		if (it->collisions) hipFree(it->collisions);
		it->collisions = (PointTriangleConstraint*)allocateCUDABuffer(sizeof(PointTriangleConstraint) * constraints.size());
		it->nCollisions = constraints.size();
		hipMemcpy(it->collisions, &constraints[0],
				sizeof(PointTriangleConstraint) * constraints.size(),
				hipMemcpyHostToDevice);
		bytes += sizeof(PointTriangleConstraint) * constraints.size();
	}
	DBG("allocated constraints %d, bytes %d", total, bytes);
	return true;
}
#endif

HIPContext::HIPContext(softbodyList_t *bodies)
{
	mSolverSteps = DEFAULT_SOLVER_STEPS;

	if (!InitDevice()) {
		ERR("CUDA Device initialization failed!");
		return;
	}

	FOREACH(it, bodies) {
		if (!*it) continue;

		if (!InitSoftBody(*it)) {
			ShutdownDevice();
			return;
		}
	}

#if 0
	if (!InitDymmyBodyCollisionConstraint()) {
		ERR("Unable to allocate collision constraints on device!");
		ShutdownDevice();
		return;
	}
#endif 
}

HIPContext::~HIPContext()
{
	ShutdownDevice();
}

void HIPContext::UpdateVertexBuffers(bool async)
{
	hipError_t err;
	glm::vec3 *ptr;
	int threadsPerBlock = 128;

	// map all in one call
	err = hipGraphicsMapResources(mResArray.size(), &mResArray[0]);
	if (err != hipSuccess) return;

	FOREACH(it, &mDescriptors) {
		size_t size;
		err = hipGraphicsResourceGetMappedPointer((void**)&ptr, &size, it->graphics);
		if (err != hipSuccess) {
			ERR("Unable to map VBO pointer");
			return;
		}
		int blockCount = it->nMapping / threadsPerBlock + 1;
		cudaUpdateVertexBufferKernel<<<blockCount, threadsPerBlock >>>(
				ptr, mPositions.data(), mMapping.data(), it->baseIdx, 
				it->mappingIdx, it->nMapping);
	}

	hipGraphicsUnmapResources(mResArray.size(), &mResArray[0]);
}

CUDASoftBodySolver::CUDASoftBodySolver(void)
	:
		mContext(0),
		mInitialized(false)
{
}

CUDASoftBodySolver::~CUDASoftBodySolver(void)
{
	if (mContext) delete mContext;
}

bool CUDASoftBodySolver::Initialize(void)
{
	if (mInitialized) return true;

	mContext = new HIPContext(&mBodies);
		if (!mContext) {
		ERR("Unable to create CUDA context.");
		return false;
	}

	mInitialized = true;
	return true;
}

void CUDASoftBodySolver::Shutdown(void)
{
	if (!mInitialized) return;

	if (mContext) {
		delete mContext;
		mContext = NULL;
	}
	mInitialized = false;
	SoftBodySolver::Shutdown();
}

void CUDASoftBodySolver::UpdateVertexBuffers(void)
{
	if (mInitialized)
		mContext->UpdateVertexBuffers(false);
}

void HIPContext::ProjectSystem(glm::float_t dt, CUDASoftBodySolver::SoftBodyWorldParameters &world)
{
	int threadsPerBlock = 128;
	int blockCount;

	if (!mPositions.size()) return;

	// predict motion
	blockCount = mPositions.size() / threadsPerBlock + 1;
	cudaProjectPositionsAndVelocitiesKernel<<<blockCount,
		threadsPerBlock>>>(world.gravity, mPositions.data(),
			mProjections.data(), mVelocities.data(), NULL,
			mInvMasses.data(), dt, mPositions.size());

	// solver
	blockCount = mPositions.size() / threadsPerBlock + 1;
	solveShapeMatchingConstraints1<<<blockCount, threadsPerBlock>>>(
			mParticlesInfo.data(),
			mRegions.data(),
			mRegionsMembersOffsets.data(),
			mShapeInitialPositions.data(),
			mProjections.data(),
			mInvMasses.data(),
			mRegionsDynamicInfo.data(),
			mPositions.size()
			);

	solveShapeMatchingConstraints2<<<blockCount, threadsPerBlock>>>(
			mParticlesInfo.data(),
			mRegions.data(),
			mRegionsDynamicInfo.data(),
			mRegionsMembersOffsets.data(),
			mShapeInitialPositions.data(),
			mMembersRegionsOffsets.data(),
			mProjections.data(),
			mPositions.size()
			);

	blockCount = mTriangles.size() / threadsPerBlock + 1;
	solveVolumePreservationConstraint1<<<blockCount, threadsPerBlock>>>(
			mParticlesInfo.data(),
			mPartials.data(),
			mProjections.data(),
			mTriangles.data(),
			mTrianglesNormals.data(),
			mTriangles.size()
			);

	blockCount = mPartials.size() / threadsPerBlock + 1;
	solveVolumePreservationConstraint2<<<blockCount, threadsPerBlock>>>(
			mDescriptorsDev.data(),
			mPartials.data(),
			mPartials.size()
			);

	blockCount = mPositions.size() / threadsPerBlock + 1;
    solveVolumePreservationConstraint3<<<blockCount, threadsPerBlock>>>(
		mParticleTriangleInfo.data(),
		mDescriptorsDev.data(),
		mTrianglesNormals.data(),
		mProjections.data(),
		mParticleTriangleIndexes.data(),
		mPositions.size());

	solveGroundWallCollisionConstraints<<<blockCount, threadsPerBlock>>>(
			mProjections.data(), mInvMasses.data(),
			world.groundLevel, world.leftWall, world.rightWall,
			world.frontWall, world.backWall, mPositions.size());

#if 0
	vector<SoftBodyDescriptor> info;
	info.resize(1);
	hipMemcpy(&info[0], mDescriptorsDev.data(), 
			sizeof(SoftBodyDescriptor),
			hipMemcpyDeviceToHost);
	ERR("%f", info[0].volume);
#endif

#if 0
	vector<glm::float_t> info;
	info.resize(mPartials.size());
	hipMemcpy(&info[0], mPartials.data(), 
			sizeof(glm::float_t) * mPartials.size(),
			hipMemcpyDeviceToHost);
	float sum = 0;
	FOREACH_R(it, info) sum += *it;
	ERR("%f", sum);
#endif

#if 0
	vector<glm::mat3> info;
	info.resize(mTest.size());
	hipMemcpy(&info[0], mTest.data(),
			sizeof(glm::mat3) * mTest.size(),
			hipMemcpyDeviceToHost);
	vector<glm::vec3> info2;
	info2.resize(mTest2.size());
	hipMemcpy(&info2[0], mTest2.data(),
			sizeof(glm::vec3) * mTest2.size(),
			hipMemcpyDeviceToHost);

	REP(k, info.size()) {
		glm::mat3 R = info[k];
		glm::vec3 mc = info2[k];
		ERR("[%f %f %f]", mc[0], mc[1], mc[2]);
		ERR("[%f %f %f %f %f %f %f %f %f]", R[0][0], R[1][0], R[2][0],
				R[0][1], R[1][1], R[2][1], R[0][2], R[1][2], R[2][2]);
	}
#endif

#if 0
	vector<glm::uint> info;
	info.resize(mMembersRegionsOffsets.size());
	hipMemcpy(&info[0], mMembersRegionsOffsets.data(),
			sizeof(glm::uint) * mMembersRegionsOffsets.size(),
			hipMemcpyDeviceToHost);

	FOREACH_R(i, info) {
		ERR("%d,", *i);
	}
#endif

	// integrate motion
	threadsPerBlock = 128;
	blockCount = mPositions.size() / threadsPerBlock + 1;
	integrateMotionKernel<<<blockCount, threadsPerBlock>>>(
			dt, mPositions.data(), mProjections.data(),
			mVelocities.data(), mPositions.size());
	hipDeviceSynchronize();

}

void CUDASoftBodySolver::ProjectSystem(glm::float_t dt)
{
	if (mInitialized)
		mContext->ProjectSystem(dt, mWorldParams);
}

void CUDASoftBodySolver::AddSoftBody(SoftBody *body)
{
	mBodies.push_back(body);
	if (!mInitialized || !mContext->InitSoftBody(body))
		ERR("Failed to add SoftBody!");
}
